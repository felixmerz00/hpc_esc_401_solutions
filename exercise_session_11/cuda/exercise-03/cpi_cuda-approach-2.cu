// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" double getTime(void);

#define NBIN 1000000000  // Number of bins
#define NUM_BLOCK  (2*56)  // Number of thread blocks
#define NUM_THREAD  (2*8)  // Number of threads per block

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
	int i;
	double x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

// Main routine that executes on the host
int main(void) {
	int NUM_BLOCK_VALUES[] =	{60, 120, 180, 240, 300, 360, 420, 600};
	int NUM_THREAD_VALUES[] = {16, 32, 48, 64, 80, 96, 112, 128, 144, 160};

	for(int b = 0; b < 8; b++){
		for(int t = 0; t < 10; t++){

			dim3 dimGrid(NUM_BLOCK_VALUES[b],1,1);  // Grid dimensions
			dim3 dimBlock(NUM_THREAD_VALUES[t],1,1);  // Block dimensions
			double *sumHost, *sumDev;  // Pointer to host & device arrays
			double pi = 0;
			int tid;

			double step = 1.0/NBIN;  // Step size
			size_t size = NUM_BLOCK_VALUES[b]*NUM_THREAD_VALUES[t]*sizeof(double);  //Array memory size
			sumHost = (double *)malloc(size);  //  Allocate array on host
			hipMalloc((void **) &sumDev, size);  // Allocate array on device
				double start = getTime();

			// Initialize array in device to 0
			hipMemset(sumDev, 0, size);
			// Do calculation on device
			cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, NUM_THREAD_VALUES[t], NUM_BLOCK_VALUES[b]); // call CUDA kernel
			// Retrieve result from device and store it in host array
			hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
			for(tid=0; tid<NUM_THREAD_VALUES[t]*NUM_BLOCK_VALUES[b]; tid++)
				pi += sumHost[tid];
			pi *= step;

			// Print results
			double delta = getTime() - start;
			printf("Num blocks: %d, num threads: %d, computation time: %.4g seconds\n", NUM_BLOCK_VALUES[b], NUM_THREAD_VALUES[t], delta);

			// Cleanup
			free(sumHost);
			hipFree(sumDev);
		}
	}


	return 0;
}

